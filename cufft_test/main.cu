#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <math.h>

#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include "../utils/utils.h"
#include "../utils/matutils.h"


#include "omp.h"

__global__ void julia_kernel( int n, double *a, double *b, double *c  ){

	int i = threadIdx.y;
	int j = threadIdx.x;

	int gi = threadIdx.y + blockDim.y*blockIdx.y;
	int gj = threadIdx.x + blockDim.x*blockIdx.x;

}


int main( int argc, char**  argv  ){

	int args_needed = 1;
	if (argc < args_needed + 1 ){
		std::cout <<"Arg number error, needed: " << args_needed<< std::endl; 	
		return 0;
	}


	std::cout << "cuFFT Test - Discrete Cosine Transform" << std::endl;
	
	// CUDA Timmers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// OMP
	int ncpu = 1;
	omp_set_num_threads(ncpu);

	// Creating matrices - using two vectors
	std::cout << "PI number: "<< M_PI << std::endl;
	
	int size_m = 8;
	int size_n = 8;

	double *x_n = new double[size_m * size_n]; 
	
	double *m_line = new double[size_m];
	double *n_line = new double[size_n];

	double k = 1;
	double N = (double)size_m;
	for (int n=0; n<N; n++){
		m_line[n] = cos(2*M_PI*k*(n/ (N-1.0)));		
	}


	std::cout << "data" << std::endl;
	for (int n=0; n<N; n++){
	 	std::cout << m_line[n] << std::endl;		
	}
	
	delete x_n;
	delete m_line;
	delete n_line;


	return 0;
}





