#include "hip/hip_runtime.h"

#define _USE_MATH_DEFINES
#include <math.h>
#include <fftw3.h>

//CBLAS
extern "C"
{
	#include <cblas.h>
}


#include <iostream>
#include <stdlib.h>
#include <stdio.h>

//CuBlas
#include <hipblas.h>

#include "omp.h"

// custom headers
#include "./include/tools.h"

/*__global__ void julia_kernel( int n, double *a, double *b, double *c  ){*/

	/*int i = threadIdx.y;*/
	/*int j = threadIdx.x;*/

	/*int gi = threadIdx.y + blockDim.y*blockIdx.y;*/
	/*int gj = threadIdx.x + blockDim.x*blockIdx.x;*/

/*}*/


int main( int argc, char**  argv  ){

	int args_needed = 1;
	if (argc < args_needed + 1 ){
		std::cout <<"Arg number error, needed: " << args_needed<< std::endl; 	
		return 0;
	}


	std::cout << "cuFFT Test - Discrete Cosine Transform" << std::endl;
	
	// CUDA Timmers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	// OMP
	int ncpu = 1;
	omp_set_num_threads(ncpu);

	// Creating matrices - using two vectors
	std::cout << "PI number: "<< M_PI << std::endl;
	
	int size_m = 16;
	int size_n = 16;

	double *x_n = new double[size_m * size_n]; 
	
	double *m_line = new double[size_m];
	double *n_line = new double[size_n];

	// Fill and Print	
	fill_vector_cos(3, size_m, m_line);
	fill_vector_cos(2, size_n, n_line);
	
	print_array(m_line, size_m);
	print_array(n_line, size_n);

	// CPU - CBLAS
	cblas_dgemm(CblasRowMajor, 		// Layout
			CblasNoTrans, 		// trans a
			CblasNoTrans,		// trans b
			16,			// m
			16,			// n
			1,			// k
			1.0,			// alpha
			m_line,			// a matrix
			1,			// lda
			n_line,			// b matrix
			16,			// ldb
			0.0,			// beta
			x_n,			// c matrix
			16			// ldc
			   );	
	
	print_array(x_n, 16*16);

	// CUDA
	// CUDA Handle
	hipblasHandle_t cublasHandle;
  	hipEvent_t startcublas;
     	hipEvent_t stopcublas;
	// cublas event create
	hipEventCreate(&startcublas);
	hipEventCreate(&stopcublas);
	
	hipblasCreate(&cublasHandle)
	// Tensor cores enabled
	hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH);	

	hipEventRecord(startcublas);
	
	hipblasGemmEx(
			cublasHandle, 
			HIPBLAS_OP_N, 
			HIPBLAS_OP_N,
			MATRIX_M, 
			MATRIX_N, 
			MATRIX_K,
		       	&alpha,
			a_fp16, 
			HIP_R_16F, 
			MATRIX_M,						                
			b_fp16, 
			HIP_R_16F, 
			MATRIX_K,
			&beta, 
			c_cublas, 
			HIP_R_32F, 
			MATRIX_M,
			HIP_R_32F, 
			CUBLAS_GEMM_DFALT_TENSOR_OP);

	hipEventRecord(stopcublas);
	
	float cublasTime;
	hipEventSynchronize(stopcublas);
	hipEventElapsedTime(&cublasTime, startcublas, stopcublas);

	std::cout << "cublas took: " << cublasTime << std::endl;

	// Free data	
	delete x_n;
	delete m_line;
	delete n_line;

	// cuda free
	hipEventDestroy(startcublas);
	hipEventDestroy(stopcublas);

	/*hipFree();*/


	return 0;
}





